#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <numeric>

__global__ void sum(int *output, int chunkSize, int numThreads, int start, int stop)
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x);
    int result = 0;
    int startNum = start + (index * chunkSize);
    for (int i = startNum; i < stop && i < (startNum + chunkSize); i++) {
        result += i;
    }

    output[index] = result

    auto step_size = 1;
    int number_of_threads = numThreads;
    while (number_of_threads > 0)
    {
        if (index < number_of_threads) // still alive?
        {
            const auto first = index * step_size * 2;
            const auto second = first + step_size;
            input[first] += input[second];
        }

        step_size <<= 1;
        number_of_threads >>= 1;
    }
}

int main(int argc, char const *argv[])
{
    long start = 1;
    long stop = 6074001000;
  
    if (argc == 3)
    {
      start = atol(argv[1]);
      stop = atol(argv[2]);
    }

    int blockSize;      // The launch configurator returned block size 
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, sum, 0, 0); 

    int numThreads = minGridSize * blockSize;
    int chunkSize = (stop - start) / numThreads;

    const int size = numThreads * sizeof(int);

    int *d;

    hipMalloc(&d, size);
    hipMemset(d, 0, size);

    sum<<<1, count / 2>>>(d, chunkSize, numThreads, start, stop);

    int result;
    hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Sum is " << result << std::endl;

    hipFree(d);

    return 0;
}