#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <numeric>
#include <cmath>

typedef unsigned long long int number_type;

__global__ void sum(number_type *input)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    number_type start = input[tid * 2];
    number_type stop = input[tid * 2 + 1];

    // Reuse the input array to save the result
    input[tid * 2] = 0;
    input[tid * 2 + 1] = 0;

    for (number_type i = start; i <= stop; i++)
    {
        number_type old = input[tid * 2 + 1];
        input[tid * 2 + 1] += i;

        // Detect overflow
        if (old > input[tid * 2 + 1])
        {
            input[tid * 2]++;
        }
    }
}

int main(int argc, char const *argv[])
{
    number_type start = 1;
    number_type stop = 1000;

    if (argc == 3)
    {
        start = atol(argv[1]);
        stop = atol(argv[2]);
    }


    const number_type range_per_thread = min(stop - start, 100ull);
    const number_type threads_per_block = 256;

    number_type num_threads = (stop - start + 1) / range_per_thread;
    number_type num_blocks = std::ceil((double) num_threads / (double) threads_per_block);

    number_type size = sizeof(number_type) * num_blocks * threads_per_block * 2;
    number_type *ranges = (number_type *)malloc(size);
    number_type current = start;


    for (int i = 0; i < num_threads; i++)
    {
        unsigned long long range_per_thread = (double)(stop - current) / (double)(num_threads - i);

        unsigned long long end = current + range_per_thread;
        ranges[i * 2] = current;
        ranges[i * 2 + 1] = end;

        current = end + 1;
    }

    number_type *ranges_gpu;
    hipMalloc(&ranges_gpu, size);
    hipMemcpy(ranges_gpu, ranges, size, hipMemcpyHostToDevice);

    sum<<<num_blocks, threads_per_block>>>(ranges_gpu);

    /*
    hipError_t err = hipGetLastError();
    const char* error = hipGetErrorString(err);
    printf("%s\n", error);
    */

    hipMemcpy(ranges, ranges_gpu, size, hipMemcpyDeviceToHost);


    __uint128_t result = 0;

    for (number_type i = 0; i < num_threads; i++)
    {
        __uint128_t upper = ranges[i * 2];
        __uint128_t lower = ranges[i * 2 + 1];

        upper <<= 64;

        result += upper;
        result += lower;
    }

    char arr[39];
    char basechars[] = "0123456789";
    int length = 0;
    while (result != 0)
    {
        int radix;
        radix = result % 10;
        result = result / 10;
        arr[length++] = basechars[radix];
    }
    if (length == 0)
    {
        std::cout << 0 << std::endl;
    }
    else
    {
        while (length--)
            std::cout << arr[length];

        std::cout << std::endl;
    }

    hipFree(ranges_gpu);

    return 0;
}