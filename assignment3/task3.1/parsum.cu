#include "hip/hip_runtime.h"


#include <iostream>
#include <numeric>
#include <math.h>

__global__ void sum(long *output, long chunkSize, int numThreads, long start, long stop)
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x);
    long result = 0;
    long startNum = start + (index * chunkSize);

    for (long i = startNum; i <= stop && i < (startNum + chunkSize); i++) {
        result += i;
    }

    if (result != 0) {
        //printf("startNum: %ld + chunkSize: %ld = result: %ld\n", startNum, startNum + chunkSize, result);
    }

    output[index] = result;

    __syncthreads();

    auto step_size = 1;
    int number_of_threads = numThreads;
    while (number_of_threads > 0)
    {
        if (index < number_of_threads) // still alive?
        {
            const auto first = index * step_size * 2;
            const auto second = first + step_size;
            output[first] += output[second];
        }

        step_size <<= 1;
        number_of_threads >>= 1;
        __syncthreads();
    }
}

int main(int argc, char const *argv[])
{
    long start = 1;
    long stop = 60000;
  
    if (argc == 3)
    {
      start = atol(argv[1]);
      stop = atol(argv[2]);
    }

    int blockSize;      // The launch configurator returned block size 
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, sum, 0, 0);

    int numThreads = minGridSize * blockSize;
    long chunkSize = ceil((stop - start) / (double)numThreads);

    if (chunkSize < 1) {
        chunkSize = 1;
    }

    const long size = numThreads * sizeof(long);

    long *d;

    hipMalloc(&d, size);
    hipMemset(d, 0, size);

    sum<<<minGridSize, blockSize>>>(d, chunkSize, numThreads, start, stop);

    long result;
    hipMemcpy(&result, d, sizeof(long), hipMemcpyDeviceToHost);

    std::cout << "Sum is " << result << std::endl;

    hipFree(d);

    return 0;
}