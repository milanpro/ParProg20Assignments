#include "hip/hip_runtime.h"


#include <iostream>
#include <numeric>

__global__ void sum(int *input)
{
    const int tid = threadIdx.x;

    auto step_size = 1;
    int number_of_threads = blockDim.x;

    while (number_of_threads > 0)
    {
        if (tid < number_of_threads) // still alive?
        {
            const auto fst = tid * step_size * 2;
            const auto snd = fst + step_size;
            input[fst] += input[snd];
        }

        step_size <<= 1;
        number_of_threads >>= 1;
    }
}

int main(int argc, char const *argv[])
{   
    const auto count = 8;
    const int size = count * sizeof(int);
    int h[] = {13, 27, 15, 14, 33, 2, 24, 6};

    int *d;

    hipMalloc(&d, size);
    hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    sum<<<1, count / 2>>>(d);

    int result;
    hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Sum is " << result << std::endl;

    hipFree(d);

    return 0;
}