#include "hip/hip_runtime.h"
#include <stdio.h>
#include <pthread.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <sys/sysinfo.h>
#include "csv.h"

__device__ double get_value(double *from, int x, int y, int width, int height)
{
  if ((x < 0 || x >= width) || (y < 0 || y >= height))
  {
    return 0;
  }

  return from[width * y + x];
}

double get_value_cpu(double *from, int x, int y, int width, int height)
{
  if ((x < 0 || x >= width) || (y < 0 || y >= height))
  {
    return 0;
  }

  return from[width * y + x];
}

__device__ void set_value(double *from, int x, int y, double value, int width)
{
  from[width * y + x] = value;
}

void set_value_cpu(double *from, int x, int y, double value, int width)
{
  from[width * y + x] = value;
}

void write_results(double *src, int width, int height)
{
  FILE *fp;
  fp = fopen("output.txt", "w");

  for (int y = 0; y < height; y++)
  {
    for (int x = 0; x < width; x++)
    {
      double value = get_value_cpu(src, x, y, width, height);

      if (value > 0.9)
      {
        fprintf(fp, "X");
      }
      else
      {
        char output[50];
        value += 0.09;

        snprintf(output, 50, "%f", value);
        fprintf(fp, "%c", output[2]);
      }
    }

    fprintf(fp, "\n");
  }

  fprintf(fp, "\n");

  fclose(fp);
}

void write_results_coords(const char *coords_filename, double *src, int width, int height)
{
  struct coord *coord_list = read_coords(coords_filename);

  FILE *fp;
  fp = fopen("output.txt", "w");

  struct coord *last_coord = coord_list;
  while (last_coord != NULL)
  {
    double value = get_value_cpu(src, last_coord->x, last_coord->y, width, height);

    fprintf(fp, "%.4f\n", value);
    last_coord = last_coord->next_coord;
  }

  fclose(fp);
}

__device__ void compute_point(double *src, double *dest, int x_coord, int y_coord, int width, int height)
{
  double acc = 0;

  for (int x = -1; x <= 1; x++)
  {
    for (int y = -1; y <= 1; y++)
    {
      acc += get_value(src, x_coord + x, y_coord + y, width, height);
    }
  }

  acc /= 9;

  set_value(dest, x_coord, y_coord, acc, width);
}

__global__ void worker(double *src, double *dest, int width, int height)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    compute_point(src, dest, col, row, width, height);
}

void set_hotspots(int round, struct hotspot *hotspot_list, double *src, int width)
{
  struct hotspot *last_hotspot = hotspot_list;
  while (last_hotspot != NULL)
  {
    if (round >= last_hotspot->startround && round < last_hotspot->endround)
    {
        set_value_cpu(src, last_hotspot->x, last_hotspot->y, 1.0f, width);
    }

    last_hotspot = last_hotspot->next_hotspot;
  }
}

int main(int argc, char const *argv[])
{
  int width;
  int height;

  width = 50;
  height = 20;
  int rounds = 32;
  const char *hotspots_filename = "test_hotspots.csv";
  const char *coords_filename = NULL; //"test_coords.csv";

  if (argc != 1)
  {
    width = atoi(argv[1]);
    height = atoi(argv[2]);
    rounds = atoi(argv[3]);
    hotspots_filename = argv[4];
    if (argc == 6)
    {
      coords_filename = argv[5];
    }
    else
    {
      coords_filename = NULL;
    }
  }

  dim3 blockSize = dim3(width, height);
  dim3 gridSize = dim3(1,1);

  hipStream_t stream;
  hipStreamCreate(&stream);

  double *src;
  double *dest;
  hipMallocManaged(&src, width * height * sizeof(double), hipMemAttachHost);
  hipMallocManaged(&dest, width * height * sizeof(double), hipMemAttachHost);
  hipStreamAttachMemAsync(stream, src);
  hipStreamAttachMemAsync(stream, dest);
  struct hotspot *hotspot_list = parse_hotspot_list(hotspots_filename);
  
  set_hotspots(0, hotspot_list, src, width);

  hipStreamSynchronize(stream);
  for (int i = 0; i < rounds; i++)
  {
    printf("Round: %d/%d starting...\n", i, rounds);

    worker<<< gridSize, blockSize, 0, stream >>>(src, dest, width, height);
    hipStreamSynchronize(stream);

    printf("Round: %d/%d done\n", i, rounds);

    //Swapping src into dest and the other way round
    double *temp = dest;
    dest = src;
    src = temp;

    set_hotspots(i + 1, hotspot_list, src, width);
  }

  hipStreamDestroy(stream);

  if (coords_filename != NULL)
  {
    write_results_coords(coords_filename, src, width, height);
  }
  else
  {
    write_results(src, width, height);
  }

  hipFree(src);
  hipFree(dest);

  return 0;
}
